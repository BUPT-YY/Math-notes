#include "hip/hip_runtime.h"
//Usage: set in vscode-code_runner:
// "code-runner.executorMapByFileExtension": {
//        ".cu": "cd $dir && nvcc -lcublas $fileName -o $fileNameWithoutExt && $dir$fileNameWithoutExt",
//    },
#include <iostream>
#include <random>
#include <cmath>
//#include <chrono>
#include <vector>
#include <algorithm>
#include <string>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"
#include "/usr/local/cuda-10.2/include/hipblas.h"
std::random_device rd;
std::default_random_engine e{rd()};
std::uniform_real_distribution<float> uniform(0,1);
using YYmatrix = std::vector<float>;


void generate(int size, float* vec) {
  //static std::vector<float> vec(size, 0);
  for(int i = 0; i < size; ++i) {
    vec[i] = uniform(e);
  }
  //return vec;
}

void matrixMmultCPU(  //both column-major
  float* A,  // N by K
  float* B,  // K by M
  float* C,  // N by M
    int N, int K, int M) { 
  for(int i = 0; i < N; ++i) {
    for(int j = 0; j < M; ++j) {
      float res = 0.0f;
      for(int k = 0; k < K; ++k) {
        res += A[i+ k*N] * B[k+j*K];
      }
      C[i+j*N] = res;
    }
  }
}


//A:N*K, B:K*M
const int N = 1024;
const int K = 2048;
const int M = 1024;

int main() {
    float* h_a, * h_b, * h_c;


    hipHostMalloc((void**)&h_a, sizeof(float)*N*K);
    hipHostMalloc((void**)&h_b, sizeof(float)*K*M);
    hipHostMalloc((void**)&h_c, sizeof(float)*N*M);

    generate(N*K, h_a);
    generate(K*M, h_b);

    float* d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(float)*N*K);
    hipMalloc(&d_b, sizeof(float)*K*M);
    hipMalloc(&d_c, sizeof(float)*N*M);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMemcpy(d_a,h_a,sizeof(float)*N*K,hipMemcpyHostToDevice); //数据从内存拷贝到显存
    hipMemcpy(d_b,h_b,sizeof(float)*K*M,hipMemcpyHostToDevice);

    float alpha = 2.0f, beta = 0.0f;

  float usedTime;
  hipEvent_t start_GPU, stop_GPU; hipEventCreate(&start_GPU); hipEventCreate(&stop_GPU);
  hipEventRecord(start_GPU, 0);
    hipblasSgemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        d_a, M,
        d_b, K,
        &beta,
        d_c, M
    );
  hipEventRecord(stop_GPU, 0);
  hipEventSynchronize(start_GPU);    //等待事件完成。
  hipEventSynchronize(stop_GPU);    //等待事件完成。记录之前的任务
  hipEventElapsedTime(&usedTime, start_GPU, stop_GPU);    //计算时间差
  hipEventDestroy(start_GPU);hipEventDestroy(stop_GPU);    //消除Event
  

    hipMemcpy(h_c, d_c, sizeof(float)*N*M, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    std::cout << "Used time: " << usedTime*1000.0f << "μs." << std::endl;
    std::cout << "GPU average Throughput: " << 2.0*N*M*K/usedTime/1.0e6 << "GFLOPS" << std::endl; 

  float* h_d = (float*)malloc(sizeof(float)*N*M); 

    matrixMmultCPU(h_a,h_b,h_d,N,K,M);
      float l2error = 0;
      for(int i = 0; i < N*M; ++i) {
          float temp = (h_c[i]-2.0f*h_d[i]);
          l2error += temp*temp;
      }
      l2error = sqrt(l2error);
      std::string result = (l2error < 0.05) ? ", test passed." : ", test failed."; 
      std::cout << "GPU error = " << l2error << result << std::endl;

  free(h_d);
  hipHostFree((void*)h_a);
  hipHostFree((void*)h_b);
  hipHostFree((void*)h_c);

    return 0;
}


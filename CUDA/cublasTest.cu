#include <iostream>
#include <random>
#include <cmath>
//#include <chrono>
#include <vector>
#include <algorithm>
#include <string>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"
#include "/usr/local/cuda-10.2/include/hipblas.h"
std::random_device rd;
std::default_random_engine e{rd()};
std::uniform_real_distribution<float> uniform(0,1);
using YYmatrix = std::vector<float>;


void generate(int size, float* vec) {
  //static std::vector<float> vec(size, 0);
  for(int i = 0; i < size; ++i) {
    vec[i] = uniform(e);
  }
  //return vec;
}




//A:N*K, B:K*M
const int N = 1024;
const int K = 2048;
const int M = 1024;

int main() {
    float* h_a, * h_b, * h_c;
    h_a = (float*)malloc(sizeof(float)*N*K);
    h_b = (float*)malloc(sizeof(float)*K*M);
    h_c = (float*)malloc(sizeof(float)*N*M);
    generate(N*K, h_a);
    generate(K*M, h_b);

    float* d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeof(float)*N*K);
    hipMalloc(&d_b, sizeof(float)*K*M);
    hipMalloc(&d_c, sizeof(float)*N*M);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMemcpy(d_a,h_a,sizeof(float)*N*K,hipMemcpyHostToDevice); //数据从内存拷贝到显存
    hipMemcpy(d_b,h_b,sizeof(float)*K*M,hipMemcpyHostToDevice);

    float alpha = 1.0f, beta = 0.0f;

  float usedTime;
  hipEvent_t start_GPU, stop_GPU; hipEventCreate(&start_GPU); hipEventCreate(&stop_GPU);
  hipEventRecord(start_GPU, 0);
    hipblasSgemm(
        handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_T,
        N, M, K,
        &alpha,
        d_a, K,
        d_b, M,
        &beta,
        d_c, N
    );
  hipEventRecord(stop_GPU, 0);
  hipEventSynchronize(start_GPU);    //等待事件完成。
  hipEventSynchronize(stop_GPU);    //等待事件完成。记录之前的任务
  hipEventElapsedTime(&usedTime, start_GPU, stop_GPU);    //计算时间差
  hipEventDestroy(start_GPU);hipEventDestroy(stop_GPU);    //消除Event
  

    hipMemcpy(h_c, d_c, sizeof(float)*N*M, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    std::cout << "Used time: " << usedTime*1000.0f << "μs." << std::endl;
    std::cout << "GPU average Throughput: " << 2.0*N*M*K/usedTime/1.0e6 << "GFLOPS" << std::endl; 

    return 0;
}


#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cmath>
//#include <chrono>
#include <vector>
#include <algorithm>
#include <string>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"

std::random_device rd;
std::default_random_engine e{rd()};
std::uniform_real_distribution<float> uniform(0,1);
using YYmatrix = std::vector<float>;


std::vector<float> generate(int size) {
  static std::vector<float> vec(size, 0);
  for(int i = 0; i < size; ++i) {
    vec[i] = uniform(e);
  }
  return vec;
}

#define TS 64
#define WPT 4
#define RTS 16   //RTS = TS/WPT
// Kernel function to add the elements of two arrays
__global__ void mmult(float* A, float* B, float* C, int K, int M) {
  
  const int col = threadIdx.x,row = threadIdx.y;
  const int globalRow = TS*blockIdx.y + row;
  const int globalCol = TS*blockIdx.x + col;
  //int blockCol = blockIdx.x, blockRow = blockIdx.y;
  
  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];
  
  float acc[WPT] = {0.0f};

  int numTiles = K/TS;
  for(int t = 0; t < numTiles; ++t) {
    for(int w = 0; w < WPT; ++w) {
      Asub[row][col+w*RTS] = A[K * globalRow + TS * t + col+w*RTS];
      Bsub[row][col+w*RTS] = B[M * (TS*t + row) + globalCol+w*RTS];
    }
    
    __syncthreads();

    for(int k = 0; k < TS; ++k)
      for(int w = 0; w < WPT; ++w)
        acc[w] += Asub[row][k] * Bsub[k][col+w*RTS];
    __syncthreads();
  }

  for(int w = 0; w < WPT; ++w)
    C[M * globalRow + globalCol + w*RTS] = acc[w];
}

void matrixMmultCPU(  //both row-major
  const YYmatrix& A,  // N by K
  const YYmatrix& B,  // K by M
        YYmatrix& C,  // N by M
        int K) {
   int N = A.size()/K, M = B.size()/K;  
  for(int i = 0; i < N; ++i) {
    for(int j = 0; j < M; ++j) {
      float res = 0.0f;
      for(int k = 0; k < K; ++k) {
        res += A[i*K + k] * B[k*M+j];
      }
      C[i*M+j] = res;
    }
  }
}

double matrixMmultGPU(  //both row-major
    YYmatrix& A,  // N by K
    YYmatrix& B,  // K by M
    YYmatrix& C,  // N by M
        int K) {
  int N = A.size()/K, M = B.size()/K;  
  float* gpu_a = nullptr, *gpu_b = nullptr, *gpu_c = nullptr; 
  hipMalloc((void**)&gpu_a, N*K*sizeof(float));
  hipMalloc((void**)&gpu_b, K*M*sizeof(float));
  hipMalloc((void**)&gpu_c, N*M*sizeof(float));


  //auto timeStart = std::chrono::system_clock::now();
  hipMemcpy(gpu_a,A.data() , N*K*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b,B.data() , K*M*sizeof(float), hipMemcpyHostToDevice);  

  float usedTime;
  hipEvent_t start_GPU, stop_GPU; hipEventCreate(&start_GPU); hipEventCreate(&stop_GPU);
  hipEventRecord(start_GPU, 0);

  dim3 threadDim(TS/WPT, TS);
  dim3 blockDim(M/TS, N/TS);
  mmult<<<blockDim,threadDim>>>(gpu_a, gpu_b, gpu_c, K,M);
  hipEventRecord(stop_GPU, 0);
  hipEventSynchronize(start_GPU);    //等待事件完成。
  hipEventSynchronize(stop_GPU);    //等待事件完成。记录之前的任务
  hipEventElapsedTime(&usedTime, start_GPU, stop_GPU);    //计算时间差
  hipEventDestroy(start_GPU);hipEventDestroy(stop_GPU);    //消除Event
  


  hipMemcpy(C.data(), gpu_c, N*M*sizeof(float), hipMemcpyDeviceToHost);
  //auto timeUsed = std::chrono::system_clock::now() - timeStart;

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_c);
  return usedTime*1000.0f;
  //return timeUsed.count()/1000.0;
}




int main()
{
  
  //This Kernel computes C = A * B, where
  // A is a N * K matrix,
  // B is a K * M matrix

  constexpr int N = 1024;
  constexpr int K = 2048;
  constexpr int M = 1024;
  
  int TestTime = 10; std::vector<float> statistics;
  int randTest = std::uniform_int_distribution<>(0, TestTime) (e);
  for(int t = 0; t < TestTime; ++t) {
    YYmatrix A = generate(N*K), B = generate(K*M), C(N*M,0), D(N*M, 0);
    statistics.push_back(matrixMmultGPU(A,B,C,K));


    if(t==0) {  //t==randTest, exam the result only once
      matrixMmultCPU(A,B,D,K);
      float l2error = 0;
      for(int i = 0; i < N; ++i) {
        for(int j = 0; j < M; ++j) {
          float temp = (C[i*M+j]-D[i*M+j]);
          l2error += temp*temp;
        } 
      }
      l2error = sqrt(l2error);
      std::string result = (l2error < 0.05) ? ", test passed." : ", test failed."; 
      std::cout << "GPU error = " << l2error << result << std::endl;
      
     /*for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", C[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;

     for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", D[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;*/
    }
  }
  float averageTime = std::accumulate(statistics.begin(), statistics.end(), 0.0f)/TestTime;
  std::cout << "GPU average used time: " << averageTime << "μs." << std::endl;
  std::cout << "GPU average Throughput: " << 2.0*N*M*K/averageTime/1000.0 << "GFLOPS" << std::endl; 
  return 0;
}

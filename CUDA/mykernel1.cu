#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cmath>
//#include <chrono>
#include <vector>
#include <algorithm>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"

std::random_device rd;
std::default_random_engine e{rd()};
std::uniform_real_distribution<float> uniform(0,1);
using YYmatrix = std::vector<float>;


std::vector<float> generate(int size) {
  static std::vector<float> vec(size, 0);
  for(int i = 0; i < size; ++i) {
    vec[i] = uniform(e);
  }
  return vec;
}

#define BLOCK_SIZE 32
// Kernel function to add the elements of two arrays
__global__ void mmult(float* A, float* B, float* C, int K, int M) {
  int blockCol = blockIdx.x, blockRow = blockIdx.y;
  float runningSum = 0.0f;
  int col = threadIdx.x,row = threadIdx.y;
  int BK = K/BLOCK_SIZE;
  for(int bk = 0; bk < BK; ++bk) {
    float* asub = A + K * BLOCK_SIZE * blockRow + BLOCK_SIZE * bk;
    float* bsub = B + M * BLOCK_SIZE * bk + BLOCK_SIZE * blockCol;
    __shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];
    as[row][col] = asub[row*K + col];
    bs[row][col] = bsub[row*M + col];
    __syncthreads();
    for(int e = 0; e < BLOCK_SIZE; ++e) {
      runningSum += as[row][e] * bs[e][col];
    }
    __syncthreads();
  }
  float* csub = C + M * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol;
  csub[row*M + col] = runningSum;
}

void matrixMmultCPU(  //both row-major
  const YYmatrix& A,  // N by K
  const YYmatrix& B,  // K by M
        YYmatrix& C,  // N by M
        int K) {
   int N = A.size()/K, M = B.size()/K;  
  for(int i = 0; i < N; ++i) {
    for(int j = 0; j < M; ++j) {
      float res = 0.0f;
      for(int k = 0; k < K; ++k) {
        res += A[i*K + k] * B[k*M+j];
      }
      C[i*M+j] = res;
    }
  }
}

double matrixMmultGPU(  //both row-major
    YYmatrix& A,  // N by K
    YYmatrix& B,  // K by M
    YYmatrix& C,  // N by M
        int K) {
  int N = A.size()/K, M = B.size()/K;  
  float* gpu_a = nullptr, *gpu_b = nullptr, *gpu_c = nullptr; 
  hipMalloc((void**)&gpu_a, N*K*sizeof(float));
  hipMalloc((void**)&gpu_b, K*M*sizeof(float));
  hipMalloc((void**)&gpu_c, N*M*sizeof(float));


  //auto timeStart = std::chrono::system_clock::now();
  hipMemcpy(gpu_a,A.data() , N*K*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b,B.data() , K*M*sizeof(float), hipMemcpyHostToDevice);  

  float usedTime;
  hipEvent_t start_GPU, stop_GPU; hipEventCreate(&start_GPU); hipEventCreate(&stop_GPU);
  hipEventRecord(start_GPU, 0);

  dim3 threadDim(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blockDim(M/BLOCK_SIZE, N/BLOCK_SIZE);
  mmult<<<blockDim,threadDim>>>(gpu_a, gpu_b, gpu_c, K,M);
  hipEventRecord(stop_GPU, 0);
  hipEventSynchronize(start_GPU);    //等待事件完成。
  hipEventSynchronize(stop_GPU);    //等待事件完成。记录之前的任务
  hipEventElapsedTime(&usedTime, start_GPU, stop_GPU);    //计算时间差
  hipEventDestroy(start_GPU);hipEventDestroy(stop_GPU);    //消除Event
  


  hipMemcpy(C.data(), gpu_c, N*M*sizeof(float), hipMemcpyDeviceToHost);
  //auto timeUsed = std::chrono::system_clock::now() - timeStart;

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_c);
  return usedTime*1000.0f;
  //return timeUsed.count()/1000.0;
}




int main(void)
{
  
  //This Kernel computes C = A * B, where
  // A is a N * K matrix,
  // B is a K * M matrix
  constexpr int N = 16*BLOCK_SIZE;
  constexpr int K = 32*BLOCK_SIZE;
  constexpr int M = 16*BLOCK_SIZE;
  
  int TestTime = 1; std::vector<float> statistics;
  for(int t = 0; t < TestTime; ++t) {
    YYmatrix A = generate(N*K), B = generate(K*M), C(N*M,0), D(N*M, 0);
    statistics.push_back(matrixMmultGPU(A,B,C,K));

    if(t==0) {  //exam the result in the first time
      matrixMmultCPU(A,B,D,K);
      float l2error = 0;
      for(int i = 0; i < N; ++i) {
        for(int j = 0; j < M; ++j) {
          float temp = (C[i*M+j]-D[i*M+j]);
          l2error += temp*temp;
        } 
      }
      l2error = sqrt(l2error);
      std::cout << "GPU error = " << l2error << std::endl;
      
     /*for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", C[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;

     for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", D[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;*/
    }
  }
  float averageTime = std::accumulate(statistics.begin(), statistics.end(), 0.0f)/TestTime;
  std::cout << "GPU average used time: " << averageTime << "μs." << std::endl;
  std::cout << "GPU average Throughput: " << 2.0*N*M*K/averageTime/1000.0 << "GFLOPS" << std::endl;



  
  return 0;
}

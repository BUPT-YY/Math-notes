#include "hip/hip_runtime.h"
//tested result:
/* My device : GTX 960M
* matrix size: 1024*2048, 2048*1024
* GPU error = 0.00692065, test passed.
* GPU average used time: 3000.61μs.
* GPU average Throughput: 178.921GFLOPS
*/

#include <iostream>
#include <random>
#include <cmath>
//#include <chrono>
#include <vector>
#include <algorithm>
#include <string>
#include "/usr/local/cuda-10.2/include/hip/hip_runtime.h"
#include "/usr/local/cuda-10.2/include/"

std::random_device rd;
std::default_random_engine e{rd()};
std::uniform_real_distribution<float> uniform(0,1);
using YYmatrix = std::vector<float>;


std::vector<float> generate(int size) {
  static std::vector<float> vec(size, 0);
  for(int i = 0; i < size; ++i) {
    vec[i] = uniform(e);
  }
  return vec;
}

#define TS 32
// Kernel function to add the elements of two arrays
__global__ void mmult(float* A, float* B, float* C, int K, int M) {
  
  const int col = threadIdx.x,row = threadIdx.y;
  const int globalRow = TS*blockIdx.y + row;
  const int globalCol = TS*blockIdx.x + col;
  //int blockCol = blockIdx.x, blockRow = blockIdx.y;
  
  __shared__ float Asub[TS][TS];
  __shared__ float Bsub[TS][TS];
  
  float acc = 0.0f;

  int numTiles = K/TS;
  for(int t = 0; t < numTiles; ++t) {

    Asub[row][col] = A[K * globalRow + TS * t + col];
    Bsub[row][col] = B[M * (TS*t + row) + globalCol];
    __syncthreads();
    for(int k = 0; k < TS; ++k) {
      acc += Asub[row][k] * Bsub[k][col];
    }
    __syncthreads();
  }
  C[M * globalRow + globalCol] = acc;
}

void matrixMmultCPU(  //both row-major
  const YYmatrix& A,  // N by K
  const YYmatrix& B,  // K by M
        YYmatrix& C,  // N by M
        int K) {
   int N = A.size()/K, M = B.size()/K;  
  for(int i = 0; i < N; ++i) {
    for(int j = 0; j < M; ++j) {
      float res = 0.0f;
      for(int k = 0; k < K; ++k) {
        res += A[i*K + k] * B[k*M+j];
      }
      C[i*M+j] = res;
    }
  }
}

double matrixMmultGPU(  //both row-major
    YYmatrix& A,  // N by K
    YYmatrix& B,  // K by M
    YYmatrix& C,  // N by M
        int K) {
  int N = A.size()/K, M = B.size()/K;  
  float* gpu_a = nullptr, *gpu_b = nullptr, *gpu_c = nullptr; 
  hipMalloc((void**)&gpu_a, N*K*sizeof(float));
  hipMalloc((void**)&gpu_b, K*M*sizeof(float));
  hipMalloc((void**)&gpu_c, N*M*sizeof(float));


  //auto timeStart = std::chrono::system_clock::now();
  hipMemcpy(gpu_a,A.data() , N*K*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_b,B.data() , K*M*sizeof(float), hipMemcpyHostToDevice);  

  float usedTime;
  hipEvent_t start_GPU, stop_GPU; hipEventCreate(&start_GPU); hipEventCreate(&stop_GPU);
  hipEventRecord(start_GPU, 0);

  dim3 threadDim(TS/WPT, TS);
  dim3 blockDim(M/TS, N/TS);
  mmult<<<blockDim,threadDim>>>(gpu_a, gpu_b, gpu_c, K,M);
  hipEventRecord(stop_GPU, 0);
  hipEventSynchronize(start_GPU);    //等待事件完成。
  hipEventSynchronize(stop_GPU);    //等待事件完成。记录之前的任务
  hipEventElapsedTime(&usedTime, start_GPU, stop_GPU);    //计算时间差
  hipEventDestroy(start_GPU);hipEventDestroy(stop_GPU);    //消除Event
  


  hipMemcpy(C.data(), gpu_c, N*M*sizeof(float), hipMemcpyDeviceToHost);
  //auto timeUsed = std::chrono::system_clock::now() - timeStart;

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_c);
  return usedTime*1000.0f;
  //return timeUsed.count()/1000.0;
}




int main()
{
  
  //This Kernel computes C = A * B, where
  // A is a N * K matrix,
  // B is a K * M matrix

  constexpr int N = 512;
  constexpr int K = 1024;
  constexpr int M = 512;
  
  int TestTime = 1; std::vector<float> statistics;
  int randTest = std::uniform_int_distribution<>(0, TestTime) (e);
  for(int t = 0; t < TestTime; ++t) {
    YYmatrix A = generate(N*K), B = generate(K*M), C(N*M,0), D(N*M, 0);
    statistics.push_back(matrixMmultGPU(A,B,C,K));

    if(t==0) {  //exam the result only once
      matrixMmultCPU(A,B,D,K);
      float l2error = 0;
      for(int i = 0; i < N; ++i) {
        for(int j = 0; j < M; ++j) {
          float temp = (C[i*M+j]-D[i*M+j]);
          l2error += temp*temp;
        } 
      }
      l2error = sqrt(l2error);
      std::string result = (l2error < 0.05) ? ", test passed." : ", test failed."; 
      std::cout << "GPU error = " << l2error << result << std::endl;
      
     /*for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", C[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;

     for(int i = 0; i < 4; ++i) {
     for(int j = 0; j < 10; ++j)
      printf("%.2f, ", D[i*M+j]);
      std::cout << std::endl;
     }std::cout << std::endl;*/
    }
  }
  float averageTime = std::accumulate(statistics.begin(), statistics.end(), 0.0f)/TestTime;
  std::cout << "GPU average used time: " << averageTime << "μs." << std::endl;
  std::cout << "GPU average Throughput: " << 2.0*N*M*K/averageTime/1000.0 << "GFLOPS" << std::endl; 
  return 0;
}
